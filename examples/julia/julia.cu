#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_complex.h>

__global__ void julia(uchar3* ptr, int width, int height, float scale, int max_iter, hipComplex c) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;
    
    if (x < width && y < height) {
        float jx = scale * (float)(width/2 - x) / (width/2);
        float jy = scale * (float)(height/2 - y) / (height/2);
        hipComplex a = make_hipComplex(jx, jy);

        int i = 0;
        for (; i < max_iter; ++i) {
            a = hipCaddf(hipCmulf(a, a), c);
            if (hipCabsf(a) > 10.0f) {
                break;
            }
        }

        int flag = (i == max_iter) ? 1 : 0;

        ptr[offset] = make_uchar3(255 * flag, 0, 0);
    }   
}



void cuda_draw_julia(uchar3* ptr, int width, int height, float scale, int max_iter, float c_x, float c_y) {
    dim3 block_dim(1);
    dim3 grid_dim(width, height);

    hipComplex c = make_hipComplex(c_x, c_y);
    julia<<<grid_dim, block_dim>>>(ptr, width, height, scale, max_iter, c);
}