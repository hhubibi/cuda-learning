#include <hip/hip_runtime.h>
#include <stdio.h>

float cuda_malloc_test(int sz, bool up) {
    int *a, *dev_a;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    a = (int*)malloc(sz * sizeof(int));
    hipMalloc((void**)&dev_a, sz*sizeof(int));

    hipEventRecord(start, 0);
    for (int i = 0; i < 100; ++i) {
        if (up) {
            hipMemcpy(dev_a, a, sz * sizeof(int), hipMemcpyHostToDevice);
        } else {
            hipMemcpy(a, dev_a, sz * sizeof(int), hipMemcpyDeviceToHost);
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_a);
    free(a);

    return elapsed;
}

float cuda_host_alloc_test(int sz, bool up) {
    int *a, *dev_a;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipHostAlloc((void**)&a, sz*sizeof(int), hipHostMallocDefault);
    hipMalloc((void**)&dev_a, sz*sizeof(int));

    hipEventRecord(start, 0);
    for (int i = 0; i < 100; ++i) {
        if (up) {
            hipMemcpy(dev_a, a, sz * sizeof(int), hipMemcpyHostToDevice);
        } else {
            hipMemcpy(a, dev_a, sz * sizeof(int), hipMemcpyDeviceToHost);
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_a);
    hipHostFree(a);

    return elapsed;
}
