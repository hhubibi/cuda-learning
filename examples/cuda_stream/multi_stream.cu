#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multi_stream_kernel(int N, int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

void multi_stream(int stream_num) {
    int N = 1024*1024;
    int FULL_DATA_SIZE = N*20;

    hipDeviceProp_t prop;
    int device_id;
    hipGetDevice(&device_id);
    hipGetDeviceProperties(&prop, device_id);
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return;
    }

    hipStream_t streams[stream_num];
    for (int i = 0; i < stream_num; ++i) {
        hipStreamCreate(&streams[i]);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, FULL_DATA_SIZE*sizeof(int));
    hipMalloc((void**)&dev_b, FULL_DATA_SIZE*sizeof(int));
    hipMalloc((void**)&dev_c, FULL_DATA_SIZE*sizeof(int));

    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; ++i) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    hipEventRecord(start, 0);

    int offset = 0;
    for (int i = 0; i < FULL_DATA_SIZE; i += N*stream_num) {
        for (int j = 0; j < stream_num; ++j) {
            offset = i + j*N;
            if (offset >= FULL_DATA_SIZE) {
                break;
            }
            hipMemcpyAsync(dev_a+offset, host_a+offset, N*sizeof(int), hipMemcpyHostToDevice, streams[j]);
            hipMemcpyAsync(dev_b+offset, host_b+offset, N*sizeof(int), hipMemcpyHostToDevice, streams[j]);
            multi_stream_kernel<<<N/256, 256, 0, streams[j]>>>(N, dev_a+offset, dev_b+offset, dev_c+offset);
            hipMemcpyAsync(host_c+offset, dev_c+offset, N*sizeof(int), hipMemcpyDeviceToHost, streams[j]);
        }
    }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf( "Time taken with %d stream:  %3.1f ms\n", stream_num, elapsed);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    
    for (int i = 0; i < stream_num; ++i) {
        hipStreamDestroy(streams[i]);
    }

}