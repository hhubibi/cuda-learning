#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void single_stream_kernel(int N, int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

void single_stream() {
    int N = 1024*1024;
    int FULL_DATA_SIZE = N*20;

    hipDeviceProp_t prop;
    int device_id;
    hipGetDevice(&device_id);
    hipGetDeviceProperties(&prop, device_id);
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipStream_t stream;
    hipStreamCreate(&stream);

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMalloc((void**)&dev_b, N*sizeof(int));
    hipMalloc((void**)&dev_c, N*sizeof(int));

    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; ++i) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    hipEventRecord(start, 0);

    for (int i = 0; i < FULL_DATA_SIZE; i += N) {
        hipMemcpyAsync(dev_a, host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream);
        single_stream_kernel<<<N/256, 256, 0, stream>>>(N, dev_a, dev_b, dev_c);
        hipMemcpyAsync(host_c+i, dev_c, N*sizeof(int), hipMemcpyDeviceToHost, stream);
    }
    hipStreamSynchronize(stream);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf( "Time taken with single stream:  %3.1f ms\n", elapsed);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);

    hipStreamDestroy(stream);
}