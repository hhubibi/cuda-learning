#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void double_stream_correct_kernel(int N, int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

void double_stream_correct() {
    int N = 1024*1024;
    int FULL_DATA_SIZE = N*20;

    hipDeviceProp_t prop;
    int device_id;
    hipGetDevice(&device_id);
    hipGetDeviceProperties(&prop, device_id);
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0;
    int *dev_a1, *dev_b1, *dev_c1;

    hipMalloc((void**)&dev_a0, N*sizeof(int));
    hipMalloc((void**)&dev_b0, N*sizeof(int));
    hipMalloc((void**)&dev_c0, N*sizeof(int));
    hipMalloc((void**)&dev_a1, N*sizeof(int));
    hipMalloc((void**)&dev_b1, N*sizeof(int));
    hipMalloc((void**)&dev_c1, N*sizeof(int));

    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; ++i) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    hipEventRecord(start, 0);

    for (int i = 0; i < FULL_DATA_SIZE; i += N*2) {
        hipMemcpyAsync(dev_a0, host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_a1, host_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1);

        hipMemcpyAsync(dev_b0, host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b1, host_b+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1);

        double_stream_correct_kernel<<<N/256, 256, 0, stream0>>>(N, dev_a0, dev_b0, dev_c0);
        double_stream_correct_kernel<<<N/256, 256, 0, stream1>>>(N, dev_a1, dev_b1, dev_c1);
        
        hipMemcpyAsync(host_c+i, dev_c0, N*sizeof(int), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(host_c+i+N, dev_c1, N*sizeof(int), hipMemcpyDeviceToHost, stream1);
    }
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf( "Time taken with double stream correct:  %3.1f ms\n", elapsed);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_a0);
    hipFree(dev_b0);
    hipFree(dev_c0);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_c1);

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);

}