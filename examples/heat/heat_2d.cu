#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "helper.cuh"

#include <hip/hip_texture_types.h>


__global__ void tex2d_copy_const_kernel(float* iptr, hipTextureObject_t tex) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = c = tex2D<float>(tex, x, y);
    if (c != 0) iptr[offset] = c;
}

__global__ void tex2d_blend_kernel(float* dst, hipTextureObject_t tex) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float t, l, c, r, b;
    t = tex2D<float>(tex, x, y - 1);
    l = tex2D<float>(tex, x - 1, y);
    c = tex2D<float>(tex, x, y);
    r = tex2D<float>(tex, x + 1, y);
    b = tex2D<float>(tex, x, y + 1);

    dst[offset] = c + SPEED * (t + b + r + l - c*4);
}

void create_tex2d_obj(hipTextureObject_t *tex, float* ptr, int size_in_bytes) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = ptr;
    resDesc.res.pitch2D.desc.f = hipChannelFormatKindFloat;
    resDesc.res.pitch2D.desc.x = 32; // bits per channel
    resDesc.res.pitch2D.width = DIM;
    resDesc.res.pitch2D.height = DIM;
    resDesc.res.pitch2D.pitchInBytes = DIM*sizeof(float);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;

    // create texture object: we only have to do this once!
    hipCreateTextureObject(tex, &resDesc, &texDesc, nullptr);
}

void tex2d_init(DataBlock* d) {
    hipEventCreate(&d->start);
    hipEventCreate(&d->end);

    int image_size = DIM*DIM*4;

    hipMalloc((void**)&d->in_ptr, image_size);
    hipMalloc((void**)&d->out_ptr, image_size);
    hipMalloc((void**)&d->const_ptr, image_size);

    create_tex2d_obj(&d->tex_const, d->const_ptr, image_size);
    create_tex2d_obj(&d->tex_in, d->in_ptr, image_size);
    create_tex2d_obj(&d->tex_out, d->out_ptr, image_size);

    float *temp = (float*)malloc( image_size );
    for (int i = 0; i < DIM*DIM; ++i) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM*100 + 100] = (MAX_TEMP + MIN_TEMP)/2;
    temp[DIM*700 + 100] = MIN_TEMP;
    temp[DIM*300 + 300] = MIN_TEMP;
    temp[DIM*200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; y++) {
        for (int x = 400; x < 500; x++) {
            temp[x + y*DIM] = MIN_TEMP;
        }
    }
    hipMemcpy(d->const_ptr, temp, image_size, hipMemcpyHostToDevice);

    for (int y = 800; y < DIM; y++) {
        for (int x = 0; x < 200; x++) {
            temp[x + y*DIM] = MAX_TEMP;
        }
    }
    hipMemcpy(d->in_ptr, temp, image_size, hipMemcpyHostToDevice);

    free(temp);

    printf("init success\n");
}

void tex2d_destroy(DataBlock* d) {
    hipDestroyTextureObject(d->tex_const);
    hipDestroyTextureObject(d->tex_in);
    hipDestroyTextureObject(d->tex_out);

    hipFree(d->in_ptr);
    hipFree(d->out_ptr);
    hipFree(d->const_ptr);

    hipEventDestroy(d->start);
    hipEventDestroy(d->end);

    printf("destroy success\n");
}


void tex2d_draw(DataBlock* d) {
    hipEventRecord(d->start);

    dim3 block_dim(16, 16);
    dim3 grid_dim(DIM/16, DIM/16);

    volatile bool dst_out = true;

    for (int i = 0; i < 90; ++i) {
        float *in, *out;
        hipTextureObject_t tex;
        if (dst_out) {
            in = d->in_ptr;
            out = d->out_ptr;
            tex = d->tex_out;
        } else {
            out = d->in_ptr;
            in = d->out_ptr;
            tex = d->tex_in;
        }
        tex2d_copy_const_kernel<<<grid_dim, block_dim>>>(in, d->tex_const);
        tex2d_blend_kernel<<<grid_dim, block_dim>>>(out, tex);
        dst_out = !dst_out;
    }

    float_to_color<<<grid_dim, block_dim>>>(d->bitmap_ptr, d->in_ptr);

    hipEventRecord(d->end);
    hipEventSynchronize(d->end);

    float elapsed;
    hipEventElapsedTime(&elapsed, d->start, d->end);
    d->total_time += elapsed;
    ++d->frames;

    printf("Average time per frame: %.f ms\n", d->total_time / d->frames);
}