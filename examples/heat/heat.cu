#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "helper.cuh"

#include <hip/hip_texture_types.h>


__global__ void tex1d_copy_const_kernel(float* iptr, hipTextureObject_t tex) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = c = tex1Dfetch<float>(tex, offset);
    if (c != 0) iptr[offset] = c;
}

__global__ void tex1d_blend_kernel(float* dst, hipTextureObject_t tex) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int offset = x + y * blockDim.x * gridDim.x;

    int left = offset -1;
    int right = offset + 1;
    if (x == 0) left++;
    if (x == DIM - 1) right--;

    int top = offset - DIM;
    int bottom = offset + DIM;
    if (y == 0) top += DIM;
    if (y == DIM - 1) bottom -= DIM;

    float t, l, c, r, b;
    t = tex1Dfetch<float>(tex, top);
    l = tex1Dfetch<float>(tex, left);
    c = tex1Dfetch<float>(tex, offset);
    r = tex1Dfetch<float>(tex, right);
    b = tex1Dfetch<float>(tex, bottom);

    dst[offset] = c + SPEED * (t + b + r + l - c*4);
}

void create_tex_obj(hipTextureObject_t *tex, float* ptr, int size_in_bytes) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = ptr;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = size_in_bytes;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;

    // create texture object: we only have to do this once!
    hipCreateTextureObject(tex, &resDesc, &texDesc, nullptr);
}

void tex1d_init(DataBlock* d) {
    hipEventCreate(&d->start);
    hipEventCreate(&d->end);

    int image_size = DIM*DIM*4;

    hipMalloc((void**)&d->in_ptr, image_size);
    hipMalloc((void**)&d->out_ptr, image_size);
    hipMalloc((void**)&d->const_ptr, image_size);

    create_tex_obj(&d->tex_const, d->const_ptr, image_size);
    create_tex_obj(&d->tex_in, d->in_ptr, image_size);
    create_tex_obj(&d->tex_out, d->out_ptr, image_size);

    float *temp = (float*)malloc( image_size );
    for (int i = 0; i < DIM*DIM; ++i) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM*100 + 100] = (MAX_TEMP + MIN_TEMP)/2;
    temp[DIM*700 + 100] = MIN_TEMP;
    temp[DIM*300 + 300] = MIN_TEMP;
    temp[DIM*200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; y++) {
        for (int x = 400; x < 500; x++) {
            temp[x + y*DIM] = MIN_TEMP;
        }
    }
    hipMemcpy(d->const_ptr, temp, image_size, hipMemcpyHostToDevice);

    for (int y = 800; y < DIM; y++) {
        for (int x = 0; x < 200; x++) {
            temp[x + y*DIM] = MAX_TEMP;
        }
    }
    hipMemcpy(d->in_ptr, temp, image_size, hipMemcpyHostToDevice);

    free(temp);

    printf("init success\n");
}

void tex1d_destroy(DataBlock* d) {
    hipDestroyTextureObject(d->tex_const);
    hipDestroyTextureObject(d->tex_in);
    hipDestroyTextureObject(d->tex_out);

    hipFree(d->in_ptr);
    hipFree(d->out_ptr);
    hipFree(d->const_ptr);

    hipEventDestroy(d->start);
    hipEventDestroy(d->end);

    printf("destroy success\n");
}


void tex1d_draw(DataBlock* d) {
    hipEventRecord(d->start);

    dim3 block_dim(16, 16);
    dim3 grid_dim(DIM/16, DIM/16);

    volatile bool dst_out = true;

    for (int i = 0; i < 90; ++i) {
        float *in, *out;
        hipTextureObject_t tex;
        if (dst_out) {
            in = d->in_ptr;
            out = d->out_ptr;
            tex = d->tex_out;
        } else {
            out = d->in_ptr;
            in = d->out_ptr;
            tex = d->tex_in;
        }
        tex1d_copy_const_kernel<<<grid_dim, block_dim>>>(in, d->tex_const);
        tex1d_blend_kernel<<<grid_dim, block_dim>>>(out, tex);
        dst_out = !dst_out;
    }

    float_to_color<<<grid_dim, block_dim>>>(d->bitmap_ptr, d->in_ptr);

    hipEventRecord(d->end);
    hipEventSynchronize(d->end);

    float elapsed;
    hipEventElapsedTime(&elapsed, d->start, d->end);
    d->total_time += elapsed;
    ++d->frames;

    printf("Average time per frame: %.f ms\n", d->total_time / d->frames);
}